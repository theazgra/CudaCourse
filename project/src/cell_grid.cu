#include "hip/hip_runtime.h"
#include <cell_grid.cuh>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>

////////////////////////////// Fitness texture ////////////////////////////////
struct TextureInfo
{
    byte *device_data;
    size_t pitch;
    hipChannelFormatDesc textureCFD;
};
// texture<unsigned char, 2, hipReadModeElementType> fitnessTexRef;
texture<uint16_t, 2, hipReadModeElementType> fitnessTexRef;
TextureInfo fitnessTex = {};
///////////////////////////////////////////////////////////////////////////////

// Number of random status must be <= 200, because only 200 state params are prepared by nvidia, more params are definitely possible
// but the user must generate them. 14*14*1*1 kernel will use 196 states which is as much as possible.
constexpr uint rngGridDim = 14;
constexpr uint rngBlockDim = 1;

struct RandomGeneratorInfo
{
    hiprandStateMtgp32 *state;
    int xMin;
    int yMin;
    int xMax;
    int yMax;
};

//////////////////////////////////////////////////////// KERNELS /////////////////////////////////////////////////////////////////////

__global__ void generate_random_population(CellGridInfo gridInfo, RandomGeneratorInfo rng)
{
    uint tIdX = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint tIdY = (blockIdx.y * blockDim.y) + threadIdx.y;
    uint strideX = blockDim.x * gridDim.x;
    uint strideY = blockDim.y * gridDim.y;

    uint rngStateOffset = (tIdX * rngGridDim) + tIdY;

    while (tIdX < gridInfo.width)
    {
        tIdY = (blockIdx.y * blockDim.y) + threadIdx.y;
        while (tIdY < gridInfo.height)
        {
            float f1 = hiprand_uniform(&rng.state[rngStateOffset]);
            float f2 = hiprand_uniform(&rng.state[rngStateOffset]);
            int x = rng.xMin + ((int)(f1 * (rng.xMax - rng.xMin) + 0.999999));
            int y = rng.yMin + ((int)(f2 * (rng.yMax - rng.yMin) + 0.999999));

            Cell rnd(x, y);
            rnd.fitness = tex2D<uint16_t>(fitnessTexRef, x, y);

            gridInfo.data[(tIdY * gridInfo.width) + tIdX] = rnd;

            tIdY += strideY;
        }
        tIdX += strideX;
    }
}

// This kernel will evole current population into new one.
template <NeighborhoodType neigh>
__global__ void evolve_kernel(const CellGridInfo currPop, CellGridInfo nextPop)
{

    uint tIdX = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint tIdY = (blockIdx.y * blockDim.y) + threadIdx.y;
    uint strideX = blockDim.x * gridDim.x;
    uint strideY = blockDim.y * gridDim.y;

    while (tIdX < currPop.width)
    {
        tIdY = (blockIdx.y * blockDim.y) + threadIdx.y;
        while (tIdY < currPop.height)
        {
            //Cell *cell = ((Cell *)((char *)currPop.data + tIdY * currPop.pitch) + tIdX);
            Cell *cell = &currPop.data[(tIdY * currPop.width) + tIdX];

            // We can't find partner in cell code, becuse we don't know the fitness value.
            // We would have to do 2 iteratios of this loops. One beforehand to just setup fitness value,
            // then synchronize all threads and find the mating partner.

            Cell *partner = nullptr;
            Cell *neighArr;
            int neighSize;
            switch (neigh)
            {
            case NeighborhoodType_L5:
            {
                neighSize = 4;
                Cell neighborhood[4];
                cell->get_neighborhood<neigh>(tIdX, tIdY, currPop, neighborhood);
                neighArr = neighborhood;
            }
            break;
            case NeighborhoodType_L9:
            case NeighborhoodType_C9:
            {
                neighSize = 8;
                Cell neighborhood[8];
                cell->get_neighborhood<neigh>(tIdX, tIdY, currPop, neighborhood);
                neighArr = neighborhood;
            }
            break;
            case NeighborhoodType_C13:
            {
                neighSize = 12;
                Cell neighborhood[12];
                cell->get_neighborhood<neigh>(tIdX, tIdY, currPop, neighborhood);
                neighArr = neighborhood;
            }
            break;
            }

            float bestFitness = -1;
            {
                for (size_t i = 0; i < neighSize; i++)
                {
                    neighArr[i].fitness = (float)tex2D<uint16_t>(fitnessTexRef, neighArr[i].x, neighArr[i].y);
                    if (neighArr[i].fitness > bestFitness)
                    {
                        bestFitness = neighArr[i].fitness;
                        partner = &neighArr[i];
                    }
                }
            }

            Cell offspring = Cell(cell, partner);
            // offspring.random_mutation();
            offspring.fitness = (float)tex2D<uint16_t>(fitnessTexRef, offspring.x, offspring.y);

            //offspring.fitness = 1.0f;
            //*((Cell *)((char *)nextPop.data + tIdY * nextPop.pitch) + tIdX) = offspring;
            nextPop.data[(tIdY * nextPop.width) + tIdX] = offspring;

            tIdY += strideY;
        }
        tIdX += strideX;
    }
}

template <unsigned int blockSize>
__device__ void warp_reduce(volatile float *sData, unsigned int tId)
{
    if (blockSize >= 64)
        sData[tId] += sData[tId + 32];
    if (blockSize >= 32)
        sData[tId] += sData[tId + 16];
    if (blockSize >= 16)
        sData[tId] += sData[tId + 8];
    if (blockSize >= 8)
        sData[tId] += sData[tId + 4];
    if (blockSize >= 4)
        sData[tId] += sData[tId + 2];
    if (blockSize >= 2)
        sData[tId] += sData[tId + 1];
}

template <unsigned int blockSize>
__global__ void smart_reduce(CellGridInfo grid, unsigned int n, float *finalSum)
{
    extern __shared__ float sData[];
    unsigned int tId = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tId;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    sData[tId] = 0.0f;

    while (i < n)
    {
        sData[tId] += grid.data[i].fitness + grid.data[i + blockSize].fitness;
        i += gridSize;
    }
    __syncthreads();

    if (blockSize >= 512)
    {
        if (tId < 256)
        {
            sData[tId] += sData[tId + 256];
        }
        __syncthreads();
    }

    if (blockSize >= 256)
    {
        if (tId < 128)
        {
            sData[tId] += sData[tId + 128];
        }
        __syncthreads();
    }

    if (blockSize >= 128)
    {
        if (tId < 64)
        {
            sData[tId] += sData[tId + 64];
        }
        __syncthreads();
    }

    if (tId < 32)
        warp_reduce<blockSize>(sData, tId);

    if (tId == 0)
    {
        atomicAdd(finalSum, sData[0]);
    }
}

////////////////////////////////////////////////// END OF KERNELS /////////////////////////////////////////////////////////////////////

CellGrid::CellGrid(const size_t width, const size_t height, KernelSettings kernelSettings)
{
    this->width = width;
    this->height = height;

    this->kernelSettings = kernelSettings;
}

CellGrid::~CellGrid()
{
    // Unbind texture and release its memory.
    CUDA_CALL(hipUnbindTexture(fitnessTexRef));
    CUDA_CALL(hipFree(fitnessTex.device_data));

    // Release populations memory.
    if (device_currPopMemory != nullptr)
        hipFree(device_currPopMemory);

    if (device_nextPopMemory != nullptr)
        hipFree(device_nextPopMemory);
}

void CellGrid::create_fitness_texture(const Image &fitnessImage)
{

    uint channelCount = fitnessImage.channel_count();
    assert((channelCount == 1) ||
           (channelCount == 2) ||
           (channelCount == 4) &&
               "Cuda texture only support 1,2 or 4 sized vectors.");

    textureWidth = fitnessImage.width();
    textureHeight = fitnessImage.height();

    size_t memoryWidth = textureWidth * pixel_byte_size(fitnessImage.image_type());
    size_t memoryRowCount = textureHeight;

    CUDA_CALL(hipMallocPitch((void **)&fitnessTex.device_data, &fitnessTex.pitch, memoryWidth, memoryRowCount));
    CUDA_CALL(hipMemcpy2D(fitnessTex.device_data, fitnessTex.pitch, fitnessImage.data(), fitnessImage.pitch(), memoryWidth, memoryRowCount, hipMemcpyHostToDevice));

    fitnessTex.textureCFD = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

    fitnessTexRef.normalized = false;
    fitnessTexRef.filterMode = hipFilterModePoint;
    fitnessTexRef.addressMode[0] = hipAddressModeClamp;
    fitnessTexRef.addressMode[1] = hipAddressModeClamp;

    CUDA_CALL(hipBindTexture2D(0, &fitnessTexRef, fitnessTex.device_data, &fitnessTex.textureCFD, textureWidth, textureHeight, fitnessTex.pitch));
}

void CellGrid::initialize_grid(const Image &fitnessImage)
{
    create_fitness_texture(fitnessImage);

    // Allocate pitched memory for populations of cells.
    // CUDA_CALL(hipMallocPitch((void **)&device_currPopMemory, &currPopPitch, width * sizeof(Cell), height));
    // CUDA_CALL(hipMallocPitch((void **)&device_nextPopMemory, &nextPopPitch, width * sizeof(Cell), height));

    //NOTE: For now we are using normal un-pitched memory.
    currPopPitch = nextPopPitch = width * sizeof(Cell);
    CUDA_CALL(hipMalloc((void **)&device_currPopMemory, width * height * sizeof(Cell)));
    CUDA_CALL(hipMalloc((void **)&device_nextPopMemory, width * height * sizeof(Cell)));

    assert(currPopPitch == nextPopPitch && "Population memory pitch doesn't align!");

    hiprandStateMtgp32 *device_randomStates;
    mtgp32_kernel_params *device_kernelParams;
    size_t stateCount = rngGridDim * rngGridDim;
    assert(stateCount <= 200 && "Only 200 state params are prepared by Nvidia.");

    CUDA_CALL(hipMalloc((void **)&device_randomStates, stateCount * sizeof(hiprandStateMtgp32)));
    CUDA_CALL(hipMalloc((void **)&device_kernelParams, sizeof(mtgp32_kernel_params)));

    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, device_kernelParams));
    CURAND_CALL(hiprandMakeMTGP32KernelState(device_randomStates, mtgp32dc_params_fast_11213, device_kernelParams, stateCount, time(NULL)));

    CellGridInfo currPop = {};
    currPop.data = device_currPopMemory;
    currPop.pitch = currPopPitch;
    currPop.width = width;
    currPop.height = height;

    RandomGeneratorInfo rng = {};
    rng.xMin = 0;
    rng.yMin = 0;
    rng.xMax = textureWidth;
    rng.yMax = textureHeight;
    printf("RNG interval xMax: %u yMax: %u\n", rng.xMax, rng.yMax);
    rng.state = device_randomStates;

    CUDA_TIMED_BLOCK_START("Initial population generation");
    generate_random_population<<<dim3(rngGridDim, rngGridDim, 1), dim3(rngBlockDim, rngBlockDim, 1)>>>(currPop, rng);
    CUDA_TIMED_BLOCK_END(true);

    CUDA_CALL(hipFree(device_randomStates));
    // CUDA_CALL(hipPeekAtLastError());
    // CUDA_CALL(hipDeviceSynchronize());
    //print_cell_grid(device_currPopMemory, currPopPitch, true);
    printf("Grid initialized\n");
}

void CellGrid::print_cell_grid(const Cell *data, const size_t pitch, bool fitness) const
{
    if (device_currPopMemory == nullptr)
        return;

    Cell *tmpMemory;
    CUDA_CALL(hipHostAlloc((void **)&tmpMemory, pitch * height, hipHostMallocWriteCombined));
    CUDA_CALL(hipMemcpy(tmpMemory, data, pitch * height, hipMemcpyDeviceToHost));

    Cell *dataPtr = tmpMemory;
    for (size_t row = 0; row < height; row++)
    {
        for (size_t col = 0; col < width; col++)
        {
            if (fitness)
                printf("%2.1f ", dataPtr[col].fitness);
            else
                printf("[%i;%i] ", dataPtr[col].x, dataPtr[col].y);
        }
        printf("\n");
        dataPtr = (Cell *)(((char *)dataPtr) + pitch);
    }

    CUDA_CALL(hipHostFree(tmpMemory));
}

void CellGrid::evolve(float &evolutionTime)
{
    CellGridInfo currPop = {};
    currPop.data = device_currPopMemory;
    currPop.pitch = currPopPitch;
    currPop.width = width;
    currPop.height = height;

    CellGridInfo nextPop = {};
    nextPop.data = device_nextPopMemory;
    nextPop.pitch = nextPopPitch;
    nextPop.width = width;
    nextPop.height = height;

    CUDA_CALL(hipMemset2D(device_nextPopMemory, nextPopPitch, 5, width * sizeof(Cell), height));
    // Memory needs to be copied only if we decide to take some cells from old population.
    //CUDA_CALL(hipMemcpy2D(device_nextPopMemory, nextPopPitch, device_currPopMemory, currPopPitch, width * sizeof(Cell), height, hipMemcpyDeviceToDevice));

    CUDA_TIMED_BLOCK_START("Evolve");
    evolve_kernel<NeighborhoodType_L5><<<kernelSettings.gridDimension, kernelSettings.blockDimension>>>(currPop, nextPop);
    CUDA_TIMED_BLOCK_END(false);
    evolutionTime = elapsedTime;

    //print_cell_grid(device_currPopMemory, currPopPitch, true);
    //printf("--------------------------------------------------------------------------------\n");
    //print_cell_grid(device_nextPopMemory, nextPopPitch, true);
    //device_currPopMemory = device_nextPopMemory;

    //CUDA_CALL(hipMemcpy(device_currPopMemory, device_nextPopMemory, sizeof(Cell)*width*height, hipMemcpyDeviceToDevice));

    Cell *tmp = device_currPopMemory;
    device_currPopMemory = device_nextPopMemory;
    device_nextPopMemory = tmp;
}

float CellGrid::get_average_fitness(float &reduceTime) const
{
    unsigned int n = width * height;
    constexpr unsigned int ReduceTPB = 512;
    unsigned int numberOfBlocks = get_number_of_parts(n, ReduceTPB);
    //printf("number of blocks %u\n", numberOfBlocks);

    dim3 dimGrid = dim3(numberOfBlocks, 1, 1);
    dim3 dimBlock = dim3(ReduceTPB, 1, 1);
    unsigned int sMemSize = ReduceTPB * sizeof(float);

    CellGridInfo gridInfo = {};
    gridInfo.data = device_currPopMemory;
    gridInfo.pitch = currPopPitch;
    gridInfo.width = width;
    gridInfo.height = height;

    CUDA_TIMED_BLOCK_START("complete_smart_reduce");
    float *device_finalSum;
    CUDA_CALL(hipMalloc((void **)&device_finalSum, sizeof(float)));
    CUDA_CALL(hipMemset(device_finalSum, 0, sizeof(float)));

    smart_reduce<ReduceTPB><<<dimGrid, dimBlock, sMemSize>>>(gridInfo, n, device_finalSum);

    float sum = 0;
    CUDA_CALL(hipMemcpy(&sum, device_finalSum, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(device_finalSum));

    CUDA_CALL(hipDeviceSynchronize());
    CUDA_TIMED_BLOCK_END(false);
    reduceTime = elapsedTime;

    return sum / (float)n;
}
