#include "hip/hip_runtime.h"
#include "../../my_cuda.cu"
#include "Array2D.h"

constexpr uint desiredBlockDim = 8;
constexpr uint rowCount = 16000;
constexpr uint colCount = 16000;

__global__ void kernel_init_column_matrix(int *matrix, size_t pitch)
{
    uint xOffset = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint yOffset = (blockIdx.y * blockDim.y) + threadIdx.y;
    // uint skipX = gridDim.x * blockDim.x;
    // uint skipY = gridDim.y * blockDim.y;

    if (xOffset < colCount && yOffset < rowCount)
    {
        int *memoryRow = (int *)((char *)matrix + (xOffset * pitch));
        memoryRow[yOffset] = (xOffset * rowCount) + yOffset;
    }
    // while (xOffset < colCount && yOffset < rowCount)
    // {

    //     xOffset += skipX;
    //     yOffset += skipY;
    // }
}

__global__ void kernel_incerement(int *matrix, size_t pitch)
{
    uint xOffset = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint yOffset = (blockIdx.y * blockDim.y) + threadIdx.y;

    uint skipX = gridDim.x * blockDim.x;
    uint skipY = gridDim.y * blockDim.y;

    while (xOffset < colCount && yOffset < rowCount)
    {
        int *memoryRow = (int *)((char *)matrix + (xOffset * pitch));
        memoryRow[yOffset]++;

        xOffset += skipX;
        yOffset += skipY;
    }
}

int main(void)
{
    int *dMatrix;
    size_t pitch = 0;

    HANDLE_ERROR(hipMallocPitch((void **)&dMatrix, &pitch, rowCount * sizeof(int), colCount));

    uint gridXDim = 1 + get_number_of_parts(rowCount, desiredBlockDim);
    uint gridYDim = 1 + get_number_of_parts(colCount, desiredBlockDim);
    dim3 blockSize(desiredBlockDim, desiredBlockDim);
    dim3 gridSize(gridXDim, gridYDim);

    printf("Pitch:\t%lu \n", pitch);
    printf("Grid:\t%u x %u\n", gridXDim, gridYDim);
    printf("Block:\t%u x %u\n", desiredBlockDim, desiredBlockDim);
    printf("Matrix:\t%u x %u\n", rowCount, colCount);

    CUDA_TIMED_BLOCK_START("Column Matrix Initialization And Incement.");
    kernel_init_column_matrix<<<gridSize, blockSize>>>(dMatrix, pitch);
    // kernel_incerement<<<gridSize, blockSize>>>(dMatrix, pitch);
    HANDLE_ERROR(hipPeekAtLastError());
    CUDA_TIMED_BLOCK_END;

    size_t resultSize = colCount * rowCount * sizeof(int);
    int **result = static_cast<int **>(malloc(resultSize));
    HANDLE_ERROR(hipMemcpy2D(result, rowCount * sizeof(int), dMatrix, pitch, rowCount * sizeof(int), colCount, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(dMatrix));

#ifndef NDEBUG
    for (uint row = 0; row < rowCount; row++)
    {
        int *rowMem = (int *)(result + (int)row);
        for (uint col = 0; col < colCount; col++)
        {
            printf("%4i ", *(rowMem + col));
        }
        printf("\n");
    }
#endif

    free(result);

    return 0;
}
