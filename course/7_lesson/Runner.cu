#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// OpenGL Graphics includes
// #include <glew.h>
//#include <GL/glew.h>
#include <GL/glew.h>
#include <GL/glut.h>
// #include <freeglut.h>
#include <cudaDefs.h>
#include <imageManager.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h> // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>			  // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h> // helper functions for CUDA error check
#include <helper_gl.h>   // helper functions for CUDA/GL interop

#include "imageKernels.cuh"

#define BLOCK_DIM 8

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

//CUDA variables
unsigned int imageWidth;
unsigned int imageHeight;
unsigned int imageBPP; //Bits Per Pixel = 8, 16, 24, or 32 bit
unsigned int imagePitch;
hipGraphicsResource_t cudaPBOResource;
hipGraphicsResource_t cudaTexResource;
texture<uchar4, 2, hipReadModeElementType> cudaTexRef;
hipChannelFormatDesc cudaTexChannelDesc;
KernelSetting ks;
unsigned char someValue = 0;

//OpenGL
unsigned int pboID;
unsigned int textureID;

unsigned int viewportWidth = 1024;
unsigned int viewportHeight = 1024;

#pragma region CUDA Routines

__global__ void applyFilter(const unsigned char someValue, const unsigned int pboWidth, const unsigned int pboHeight, unsigned char *pbo)
{
	//TODO 9: Create a filter that replaces Red spectrum of RGBA pbo such that RED=someValue
	//
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int stridex = blockDim.x * gridDim.x;
	int stridey = blockDim.y * gridDim.y;

	while (tidx < pboWidth)
	{

		tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
		while (tidy < pboHeight)
		{

			//TODO: Do something with the image.

			tidy += stridey;
		}
		tidx += stridex;
	}
}

void cudaWorker()
{
	hipArray *array;

	//TODO 3: Map cudaTexResource
	hipGraphicsMapResources(1, &cudaTexResource, 0);

	//TODO 4: Get Mapped Array of cudaTexResource
	hipGraphicsSubResourceGetMappedArray(&array, cudaTexResource, 0, 0);

	//TODO 5: Get cudaTexChannelDesc from previously obtained array
	hipGetChannelDesc(&cudaTexChannelDesc, array);

	//TODO 6: Bind cudaTexRef to array
	hipBindTextureToArray(&cudaTexRef, array, &cudaTexChannelDesc);

	checkError();

	unsigned char *pboData;
	size_t pboSize;
	//TODO 7: Map cudaPBOResource
	hipGraphicsMapResources(1, &cudaPBOResource, 0);

	//TODO 7: Map Mapped pointer to cudaPBOResource data
	hipGraphicsResourceGetMappedPointer((void **)&pboData, &pboSize, cudaPBOResource);
	checkError();

	//TODO 8: Set KernelSetting variable ks (dimBlock, dimGrid, etc.) such that block will have BLOCK_DIM x BLOCK_DIM threads
	ks.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM);
	ks.dimGrid = dim3(getNumberOfParts(imageWidth, BLOCK_DIM), getNumberOfParts(imageHeight, BLOCK_DIM), 1);

	//Calling applyFileter kernel
	someValue++;
	if (someValue > 255)
		someValue = 0;
	applyFilter<<<ks.dimGrid, ks.dimBlock>>>(someValue, imageWidth, imageHeight, pboData);

	//Following code release mapped resources, unbinds texture and ensures that PBO data will be coppied into OpenGL texture. Do not modify following code!
	hipUnbindTexture(&cudaTexRef);
	hipGraphicsUnmapResources(1, &cudaPBOResource, 0);
	hipGraphicsUnmapResources(1, &cudaTexResource, 0);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);
	glBindTexture(GL_TEXTURE_2D, textureID);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageWidth, imageHeight, GL_RGBA, GL_UNSIGNED_BYTE, NULL); //Source parameter is NULL, Data is coming from a PBO, not host memory
}

void initCUDAtex()
{
	cudaGLSetGLDevice(0);
	checkError();

	//CUDA Texture settings
	cudaTexRef.normalized = false;					  //Otherwise TRUE to access with normalized texture coordinates
	cudaTexRef.filterMode = hipFilterModePoint;	  //Otherwise texRef.filterMode = hipFilterModeLinear; for Linear interpolation of texels
	cudaTexRef.addressMode[0] = hipAddressModeClamp; //No repeat texture pattern
	cudaTexRef.addressMode[1] = hipAddressModeClamp; //No repeat texture pattern

	//TODO 1: Register OpenGL texture to CUDA resource
	hipGraphicsGLRegisterImage(&cudaTexResource, textureID, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly);
	checkError();

	//TODO 2: Register PBO to CUDA resource
	hipGraphicsGLRegisterBuffer(&cudaPBOResource, pboID, hipGraphicsRegisterFlagsWriteDiscard);
	checkError();
}

void releaseCUDA()
{
	hipGraphicsUnregisterResource(cudaPBOResource);
	hipGraphicsUnregisterResource(cudaTexResource);
}
#pragma endregion

#pragma region OpenGL Routines - DO NOT MODIFY THIS SECTION !!!

void loadTexture(const char *imageFileName)
{
	FreeImage_Initialise();
	FIBITMAP *tmp = ImageManager::GenericLoader(imageFileName, 0);

	imageWidth = FreeImage_GetWidth(tmp);
	imageHeight = FreeImage_GetHeight(tmp);
	imageBPP = FreeImage_GetBPP(tmp);
	imagePitch = FreeImage_GetPitch(tmp);

	//OpenGL Texture
	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &textureID);
	glBindTexture(GL_TEXTURE_2D, textureID);

	//WARNING: Just some of inner format are supported by CUDA!!!
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, imageWidth, imageHeight, 0, GL_BGRA, GL_UNSIGNED_BYTE, FreeImage_GetBits(tmp));
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

	FreeImage_Unload(tmp);
}

void preparePBO()
{
	glGenBuffers(1, &pboID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);											   // Make this the current UNPACK buffer (OpenGL is state-based)
	glBufferData(GL_PIXEL_UNPACK_BUFFER, imageWidth * imageHeight * 4, NULL, GL_DYNAMIC_COPY); // Allocate data for the buffer. 4-channel 8-bit image
}

void my_display()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, textureID);

	//I know this is a very old OpenGL, but we want to practice CUDA :-)
	//Now it will be a wasted time to learn you current features of OpenGL. Sorry for that however, you can visit my second seminar dealing with Computer Graphics (CG2).
	glBegin(GL_QUADS);

	glTexCoord2d(0, 0);
	glVertex2d(0, 0);
	glTexCoord2d(1, 0);
	glVertex2d(viewportWidth, 0);
	glTexCoord2d(1, 1);
	glVertex2d(viewportWidth, viewportHeight);
	glTexCoord2d(0, 1);
	glVertex2d(0, viewportHeight);

	glEnd();

	glDisable(GL_TEXTURE_2D);

	glFlush();
	glutSwapBuffers();
}

void my_resize(GLsizei w, GLsizei h)
{
	viewportWidth = w;
	viewportHeight = h;

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glViewport(0, 0, viewportWidth, viewportHeight);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, viewportWidth, 0, viewportHeight);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glutPostRedisplay();
}

void my_idle()
{
	cudaWorker();
	glutPostRedisplay();
}

void initGL(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(viewportWidth, viewportHeight);
	glutInitWindowPosition(0, 0);
	glutCreateWindow(":-)");

	glutDisplayFunc(my_display);
	glutReshapeFunc(my_resize);
	glutIdleFunc(my_idle);
	glutSetCursor(GLUT_CURSOR_CROSSHAIR);

	// initialize necessary OpenGL extensions
	//glutInit()
	glewInit();
	//glutInit(nullptr, nullptr);

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glShadeModel(GL_SMOOTH);
	glViewport(0, 0, viewportWidth, viewportHeight);

	glFlush();
}

void releaseOpenGL()
{
	if (textureID > 0)
		glDeleteTextures(1, &textureID);
	if (pboID > 0)
		glDeleteBuffers(1, &pboID);
}

#pragma endregion

void releaseResources()
{
	releaseCUDA();
	releaseOpenGL();
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	initGL(argc, argv);

	loadTexture("../lena.png");

	preparePBO();

	initCUDAtex();

	//start rendering mainloop
	glutMainLoop();
	atexit(releaseResources);
}
