#include "hip/hip_runtime.h"
// includes, cuda
#include <hip/hip_runtime.h>

#include <cudaDefs.h>
#include <imageManager.h>

#include "imageKernels.cuh"

#define BLOCK_DIM 8

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

texture<float, 2, hipReadModeElementType> texRef;
hipChannelFormatDesc texChannelDesc;

unsigned char *dImageData = 0;
unsigned int imageWidth;
unsigned int imageHeight;
unsigned int imageBPP; //Bits Per Pixel = 8, 16, 24, or 32 bit
unsigned int imagePitch;

size_t texPitch;
float *dLinearPitchTextureData = 0;
hipArray *dArrayTextureData = 0;

KernelSetting ks;

float *dOutputData = 0;

void loadSourceImage(const char *imageFileName)
{
	FreeImage_Initialise();
	FIBITMAP *tmp = ImageManager::GenericLoader(imageFileName, 0);

	imageWidth = FreeImage_GetWidth(tmp);
	imageHeight = FreeImage_GetHeight(tmp);
	imageBPP = FreeImage_GetBPP(tmp);
	imagePitch = FreeImage_GetPitch(tmp);

	hipMalloc((void **)&dImageData, imagePitch * imageHeight * imageBPP / 8);
	hipMemcpy(dImageData, FreeImage_GetBits(tmp), imagePitch * imageHeight * imageBPP / 8, hipMemcpyHostToDevice);

	checkHostMatrix<unsigned char>(FreeImage_GetBits(tmp), imagePitch, imageHeight, imageWidth, "%hhu ", "Result of Linear Pitch Text");
	checkDeviceMatrix<unsigned char>(dImageData, imagePitch, imageHeight, imageWidth, "%hhu ", "Result of Linear Pitch Text");

	FreeImage_Unload(tmp);
	FreeImage_DeInitialise();
}

void createTextureFromLinearPitchMemory()
{
	//Floating Point Texture Data
	hipMallocPitch((void **)&dLinearPitchTextureData, &texPitch, imageWidth * sizeof(float), imageHeight);

	//Converts custom image data to float and stores result in the float_pitch_linear_data
	switch (imageBPP)
	{
	case 8:
		colorToFloat<8><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	case 16:
		colorToFloat<16><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	case 24:
		colorToFloat<24><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	case 32:
		colorToFloat<32><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	}

	checkDeviceMatrix<float>(dLinearPitchTextureData, texPitch, imageHeight, imageWidth, "%6.1f ", "Result of Linear Pitch Text");

	//Texture settings
	texChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	texRef.normalized = false;
	texRef.filterMode = hipFilterModePoint;
	texRef.addressMode[0] = hipAddressModeClamp;
	texRef.addressMode[1] = hipAddressModeClamp;

	hipBindTexture2D(0, &texRef, dLinearPitchTextureData, &texChannelDesc, imageWidth, imageHeight, texPitch);
}

void createTextureFrom2DArray()
{
	//Texture settings
	texChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	texRef.normalized = false;
	texRef.filterMode = hipFilterModePoint;
	texRef.addressMode[0] = hipAddressModeClamp;
	texRef.addressMode[1] = hipAddressModeClamp;

	//Converts custom image data to float and stores result in the float_linear_data
	float *dLinearTextureData = 0;
	hipMalloc((void **)&dLinearTextureData, imageWidth * imageHeight * sizeof(float));
	switch (imageBPP)
	{
	case 8:
		colorToFloat<8><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, imageWidth, dLinearTextureData);
		break;
	case 16:
		colorToFloat<16><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, imageWidth, dLinearTextureData);
		break;
	case 24:
		colorToFloat<24><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, imageWidth, dLinearTextureData);
		break;
	case 32:
		colorToFloat<32><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, imageWidth, dLinearTextureData);
		break;
	}
	hipMallocArray(&dArrayTextureData, &texChannelDesc, imageWidth, imageHeight);
	hipMemcpyToArray(dArrayTextureData, 0, 0, dLinearTextureData, imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToDevice);
	hipBindTextureToArray(&texRef, dArrayTextureData, &texChannelDesc);

	hipFree(dLinearTextureData);
}

void releaseMemory()
{
	hipUnbindTexture(texRef);
	if (dImageData != 0)
		hipFree(dImageData);
	if (dLinearPitchTextureData != 0)
		hipFree(dLinearPitchTextureData);
	if (dArrayTextureData)
		hipFreeArray(dArrayTextureData);
	if (dOutputData)
		hipFree(dOutputData);
}

__global__ void texKernel(const unsigned int texWidth, const unsigned int texHeight, float *dst)
{
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	int tx = blockIdx.x * blockDim.x + threadIdx.x;

	if ((tx < texWidth) && (ty < texHeight))
	{
		dst[ty * texWidth + tx] = tex2D(texRef, tx, ty);
	}
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	loadSourceImage("../terrain10x10.tif");

	hipMalloc((void **)&dOutputData, imageWidth * imageHeight * sizeof(float));

	ks.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	ks.blockSize = BLOCK_DIM * BLOCK_DIM;
	ks.dimGrid = dim3((imageWidth + BLOCK_DIM - 1) / BLOCK_DIM, (imageHeight + BLOCK_DIM - 1) / BLOCK_DIM, 1);

	//Test 1 - texture stored in linear pitch memory
	createTextureFromLinearPitchMemory();
	texKernel<<<ks.dimGrid, ks.dimBlock>>>(imageWidth, imageHeight, dOutputData);
	checkDeviceMatrix<float>(dOutputData, imageWidth * sizeof(float), imageHeight, imageWidth, "%6.1f ", "dOutputData");

	//Test 2 - texture stored in 2D array
	createTextureFrom2DArray();
	texKernel<<<ks.dimGrid, ks.dimBlock>>>(imageWidth, imageHeight, dOutputData);
	checkDeviceMatrix<float>(dOutputData, imageWidth * sizeof(float), imageHeight, imageWidth, "%6.1f ", "dOutputData");

	releaseMemory();
}
