#include "hip/hip_runtime.h"
// #include "../../my_cuda.cu"

#include <cudaDefs.h>
#include <imageManager.h>
#include "imageKernels.cuh"

#define BLOCK_DIM 8

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

texture<float, 2, hipReadModeElementType> texRef; // declared texture reference must be at file-scope !!!

hipChannelFormatDesc texChannelDesc;

unsigned char *dImageData = 0;
unsigned int imageWidth;
unsigned int imageHeight;
unsigned int imageBPP; //Bits Per Pixel = 8, 16, 24, or 32 bit
unsigned int imagePitch;

size_t texPitch;
float *dLinearPitchTextureData = 0;
hipArray *dArrayTextureData = 0;

KernelSetting ks;

float *dOutputData = 0;

void loadSourceImage(const char *imageFileName)
{
	FreeImage_Initialise();
	FIBITMAP *tmp = ImageManager::GenericLoader(imageFileName, 0);

	imageWidth = FreeImage_GetWidth(tmp);
	imageHeight = FreeImage_GetHeight(tmp);
	imageBPP = FreeImage_GetBPP(tmp);
	imagePitch = FreeImage_GetPitch(tmp); // FREEIMAGE align row data ... You have to use pitch instead of width

	hipMalloc((void **)&dImageData, imagePitch * imageHeight * imageBPP / 8);
	hipMemcpy(dImageData, FreeImage_GetBits(tmp), imagePitch * imageHeight * imageBPP / 8, hipMemcpyHostToDevice);

	checkHostMatrix<unsigned char>(FreeImage_GetBits(tmp), imagePitch, imageHeight, imageWidth, "%hhu ", "Result of Linear Pitch Text");
	checkDeviceMatrix<unsigned char>(dImageData, imagePitch, imageHeight, imageWidth, "%hhu ", "Result of Linear Pitch Text");

	FreeImage_Unload(tmp);
	FreeImage_DeInitialise();
}

void createTextureFromLinearPitchMemory()
{
	hipMallocPitch((void **)&dLinearPitchTextureData, &texPitch, imageWidth * sizeof(float), imageHeight);

	switch (imageBPP)
	{
	case 8:
		colorToFloat<8><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	case 16:
		colorToFloat<16><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	case 24:
		colorToFloat<24><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	case 32:
		colorToFloat<32><<<ks.dimGrid, ks.dimBlock>>>(dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData);
		break;
	default:
		break;
	}

	checkDeviceMatrix<float>(dLinearPitchTextureData, texPitch, imageHeight, imageWidth, "%6.1f ", "Result of Linear Pitch Text");

	texChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	texRef.normalized = false;
	texRef.addressMode[0] = hipAddressModeClamp;
	texRef.addressMode[1] = hipAddressModeClamp;
	texRef.filterMode = hipFilterModePoint;

	hipBindTexture2D(0, &texRef, dLinearPitchTextureData, &texChannelDesc, imageWidth, imageHeight, texPitch);
}

void createTextureFrom2DArray()
{
	//TODO: Define texture (texRef) parameters

	//TODO: Define texture channel descriptor (texChannelDesc)
	//texChannelDesc = ...

	//Converts custom image data to float and stores result in the float_linear_data
	float *dLinearTextureData = 0;
	hipMalloc((void **)&dLinearTextureData, imageWidth * imageHeight * sizeof(float));
	switch (imageBPP)
	{
		//TODO: Here call your kernel to convert image into linear memory (no pitch!!!)
	}

	hipMallocArray(&dArrayTextureData, &texChannelDesc, imageWidth, imageHeight);

	//TODO: copy data into cuda array (dArrayTextureData)
	//hipMemcpyToArray(...);

	//TODO: Bind texture
	//cudaBind...

	hipFree(dLinearTextureData);
}

void releaseMemory()
{
	hipUnbindTexture(texRef);
	if (dImageData != 0)
		hipFree(dImageData);
	if (dLinearPitchTextureData != 0)
		hipFree(dLinearPitchTextureData);
	if (dArrayTextureData)
		hipFreeArray(dArrayTextureData);
	if (dOutputData)
		hipFree(dOutputData);
}

__global__ void texKernel(const unsigned int texWidth, const unsigned int texHeight, float *dst)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	//TODO some kernel
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	loadSourceImage("terrain10x10.tif");

	hipMalloc((void **)&dOutputData, imageWidth * imageHeight * sizeof(float));

	ks.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	ks.blockSize = BLOCK_DIM * BLOCK_DIM;
	ks.dimGrid = dim3((imageWidth + BLOCK_DIM - 1) / BLOCK_DIM, (imageHeight + BLOCK_DIM - 1) / BLOCK_DIM, 1);

	//Test 1 - texture stored in linear pitch memory
	createTextureFromLinearPitchMemory();
	texKernel<<<ks.dimGrid, ks.dimBlock>>>(imageWidth, imageHeight, dOutputData);
	checkDeviceMatrix<float>(dOutputData, imageWidth * sizeof(float), imageHeight, imageWidth, "%6.1f ", "dOutputData");

	//Test 2 - texture stored in 2D array
	createTextureFrom2DArray();
	texKernel<<<ks.dimGrid, ks.dimBlock>>>(imageWidth, imageHeight, dOutputData);
	checkDeviceMatrix<float>(dOutputData, imageWidth * sizeof(float), imageHeight, imageWidth, "%6.1f ", "dOutputData");

	releaseMemory();
}
