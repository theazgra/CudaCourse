#include "hip/hip_runtime.h"
// #include <cudaDefs.h>
// #include <hipblas.h>
#include "../../my_cuda.cu"
#include <hipblas.h>
#include <hipblas.h>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

hipblasStatus_t status = hipblasStatus_t();
hipblasHandle_t handle = hipblasHandle_t();

const unsigned int N = 5;
const unsigned int dim = 3;
const unsigned int MEMSIZE = N * dim * sizeof(float);
const unsigned int THREAD_PER_BLOCK = 128;
const unsigned int GRID_SIZE = (N * dim + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

void fillData(float *data, const unsigned int length, const unsigned int dim)
{
	unsigned int id = 0;
	for (unsigned int i = 0; i < length; i++)
	{
		for (unsigned int j = 0; j < dim; j++)
		{
			data[id++] = i & 255; //=i%256
		}
	}
}

void fillDataWithNumber(float *data, const unsigned int length, const unsigned int dim, const float number)
{
	unsigned int id = 0;
	for (unsigned int i = 0; i < length; i++)
	{
		for (unsigned int j = 0; j < dim; j++)
		{
			data[id++] = number;
		}
	}
}

__global__ void kernelPowerTwo(const float *a, const float *b, const unsigned int length, float *a2, float *b2)
{
	//TODO:
}

int main(int argc, char *argv[])
{
	//initializeCUDA(deviceProp);
	status = hipblasCreate(&handle);

	float alpha, beta;
	float *a, *b, *m;
	float *da, *da2, *db, *db2, *dm;
	float *ones, *dones;

	// paged-locked allocation
	hipHostAlloc((void **)&a, MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void **)&b, MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void **)&ones, MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void **)&m, N * N * sizeof(float), hipHostMallocDefault);

	hipMalloc((void **)&da, MEMSIZE);
	hipMalloc((void **)&da2, MEMSIZE);
	hipMalloc((void **)&db, MEMSIZE);
	hipMalloc((void **)&db2, MEMSIZE);
	hipMalloc((void **)&dones, MEMSIZE);
	hipMalloc((void **)&dm, N * N * sizeof(float));

	fillData(a, N, dim);
	fillData(b, N, dim);
	fillDataWithNumber(ones, N, dim, 1.0f);

	//Copy data to DEVICE
	hipMemcpy(da, a, MEMSIZE, hipMemcpyHostToDevice);
	hipMemcpy(db, b, MEMSIZE, hipMemcpyHostToDevice);
	hipMemcpy(dones, ones, MEMSIZE, hipMemcpyHostToDevice);

	//TODO 1: Process a -> a^2  and b->b^2
	kernelPowerTwo<<<GRID_SIZE, THREAD_PER_BLOCK>>>(da, db, N * dim, da2, db2);

	//TODO 2: Process a^2 + b^2 using CUBLAS //pair-wise operation such that the result is dm[N*N] matrix

	//TODO 3: Process -2ab and sum with previous result stored in dm using CUBLAS

	// checkDeviceMatrix<float>(da, sizeof(float) * dim, N, dim, "%f ", "A");
	// checkDeviceMatrix<float>(da2, sizeof(float) * dim, N, dim, "%f ", "A^2");
	// checkDeviceMatrix<float>(db, sizeof(float) * dim, N, dim, "%f ", "B");
	// checkDeviceMatrix<float>(db2, sizeof(float) * dim, N, dim, "%f ", "B^2");
	// checkDeviceMatrix<float>(dones, sizeof(float) * dim, N, dim, "%f ", "ONES");
	// checkDeviceMatrix<float>(dm, sizeof(float) * N, N, N, "%f ", "M");

	hipFree(da);
	hipFree(da2);
	hipFree(db);
	hipFree(db2);
	hipFree(dm);
	hipFree(dones);
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(m);
	hipHostFree(ones);

	status = hipblasDestroy(handle);
}
