//#include <cuda.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <vector>

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define HANDLE_NULL(a)                                      \
    {                                                       \
        if (a == NULL)                                      \
        {                                                   \
            printf("Host memory failed in %s at line %d\n", \
                   __FILE__, __LINE__);                     \
            exit(EXIT_FAILURE);                             \
        }                                                   \
    }

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

void chapter_1_to_3()
{
    int c;
    int *device_c;

    hipMalloc((void **)&device_c, sizeof(int));
    add<<<1, 1>>>(2, 7, device_c);
    hipMemcpy(&c, device_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_c);
    printf("2 + 7 = %i\nComputed via CUDA, yaaay.\n", c);

    int cudaDeviceCount;
    hipGetDeviceCount(&cudaDeviceCount);
    printf("This computer contains %i CUDA enabled GPU.\n", cudaDeviceCount);

    std::vector<hipDeviceProp_t> deviceInfos;
    for (size_t i = 0; i < cudaDeviceCount; i++)
    {
        hipDeviceProp_t info = {};
        hipGetDeviceProperties(&info, i);
        deviceInfos.push_back(info);
        printf("Loaded info about %s\n", info.name);
    }

    // We can request device with certain capabilities like this:
    hipDeviceProp_t requirement;
    memset(&requirement, 0, sizeof(hipDeviceProp_t));
    // Request version 6.0
    requirement.major = 6;
    requirement.minor = 0;

    int returnedDevice;
    HANDLE_ERROR(hipChooseDevice(&returnedDevice, &requirement));
    printf("hipChooseDevice returned: %i\n", returnedDevice);
    HANDLE_ERROR(hipSetDevice(returnedDevice));
}

constexpr int arrSize = 1000;

__global__ void device_add(int *a, int *b, int *c)
{
    int blockId = blockIdx.x;
    if (blockId < arrSize)
        c[blockId] = a[blockId] + b[blockId];
}

void chapter_4()
{
    int a[arrSize], b[arrSize], c[arrSize];
    int *device_a, *device_b, *device_c;

    hipMalloc((void **)&device_a, arrSize * sizeof(int));
    hipMalloc((void **)&device_b, arrSize * sizeof(int));
    hipMalloc((void **)&device_c, arrSize * sizeof(int));

    for (size_t i = 0; i < arrSize; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(device_a, a, arrSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, arrSize * sizeof(int), hipMemcpyHostToDevice);

    device_add<<<arrSize, 1>>>(device_a, device_b, device_c);

    hipMemcpy(c, device_c, arrSize * sizeof(int), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < arrSize; i++)
    {
        printf("%i + %i = %i\n", a[i], b[i], c[i]);
    }
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
}

int main(void)
{
    // Chapter 1 - 3
    //chapter_1_to_3();
    chapter_4();
    return 0;
}
