//#include <cuda.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <vector>

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define HANDLE_NULL(a)                                      \
    {                                                       \
        if (a == NULL)                                      \
        {                                                   \
            printf("Host memory failed in %s at line %d\n", \
                   __FILE__, __LINE__);                     \
            exit(EXIT_FAILURE);                             \
        }                                                   \
    }

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main(void)
{
    int c;
    int *device_c;

    hipMalloc((void **)&device_c, sizeof(int));
    add<<<1, 1>>>(2, 7, device_c);
    hipMemcpy(&c, device_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_c);
    printf("2 + 7 = %i\nComputed via CUDA, yaaay.\n", c);

    int cudaDeviceCount;
    hipGetDeviceCount(&cudaDeviceCount);
    printf("This computer contains %i CUDA enabled GPU.\n", cudaDeviceCount);

    std::vector<hipDeviceProp_t> deviceInfos;
    for (size_t i = 0; i < cudaDeviceCount; i++)
    {
        hipDeviceProp_t info = {};
        hipGetDeviceProperties(&info, i);
        deviceInfos.push_back(info);
        printf("Loaded info about %s\n", info.name);
    }

    // We can request device with certain capabilities like this:
    hipDeviceProp_t requirement;
    memset(&requirement, 0, sizeof(hipDeviceProp_t));
    // Request version 6.0
    requirement.major = 6;
    requirement.minor = 0;

    int returnedDevice;
    HANDLE_ERROR(hipChooseDevice(&returnedDevice, &requirement));
    printf("hipChooseDevice returned: %i\n", returnedDevice);
    HANDLE_ERROR(hipSetDevice(returnedDevice));
    return 0;
}