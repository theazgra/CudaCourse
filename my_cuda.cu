#include <stdio.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <math.h>

typedef unsigned char byte;

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define HANDLE_NULL(a)                                      \
    {                                                       \
        if (a == NULL)                                      \
        {                                                   \
            printf("Host memory failed in %s at line %d\n", \
                   __FILE__, __LINE__);                     \
            exit(EXIT_FAILURE);                             \
        }                                                   \
    }

inline size_t get_number_of_parts(size_t whole, size_t divider)
{
    return ((whole + divider - 1) / divider);
}