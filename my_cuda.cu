#include <stdio.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <math.h>

typedef unsigned char byte;

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define HANDLE_NULL(a)                                      \
    {                                                       \
        if (a == NULL)                                      \
        {                                                   \
            printf("Host memory failed in %s at line %d\n", \
                   __FILE__, __LINE__);                     \
            exit(EXIT_FAILURE);                             \
        }                                                   \
    }

inline size_t get_number_of_parts(size_t whole, size_t divider)
{
    return ((whole + divider - 1) / divider);
}

#define CUDA_TIMED_BLOCK_START(fn_name) \
    const char *___tmdFnName = fn_name; \
    hipEvent_t startEvent, stopEvent;  \
    float elapsedTime;                  \
    hipEventCreate(&startEvent);       \
    hipEventCreate(&stopEvent);        \
    hipEventRecord(startEvent, 0);

#define CUDA_TIMED_BLOCK_END                                   \
    hipEventRecord(stopEvent, 0);                             \
    hipEventSynchronize(stopEvent);                           \
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent); \
    printf("%s took: %f ms\n", ___tmdFnName, elapsedTime);     \
    hipEventDestroy(startEvent);                              \
    hipEventDestroy(stopEvent);